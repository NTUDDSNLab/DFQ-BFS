#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <bits/stdc++.h>
#include <hip/hip_cooperative_groups.h>
using namespace std;
using namespace cooperative_groups;
namespace cg = cooperative_groups;

// global-q/scan-free/non-sep/lower-bound/

#define FTR_SIZE_S 33554432

#define TILE_LOG_S 0
#define TILE_SIZ_S 1

typedef struct
{
	int start;     // Index of first adjacent node in Ea	
	int length;    // Number of adjacent nodes 
} Node;

__device__ inline void VISIT_tiled(thread_group tile, Node *node, int *edge, unsigned int *cost, bool *done, int ftr, int *ftr_SQ, int *ftr_size_SQ)
{
    int start = node[ftr].start;
    int end = start + node[ftr].length;
    
    for(int i = start+tile.thread_rank(); i < end; i += tile.size()){
        int nid = edge[i];
        if(atomicCAS(&(cost[nid]), (unsigned int)(0xffffffff), (unsigned int)(cost[ftr] + 1)) == (unsigned int)(0xffffffff)){
            if(node[nid].length) {
                ftr_SQ[atomicAdd(ftr_size_SQ, 1)] = nid;
            }
            *done = false;
        }
    }
}

__device__ int ftr_SQ_a[FTR_SIZE_S], ftr_SQ_b[FTR_SIZE_S];
__device__ int ftr_size_SQ_a, ftr_size_SQ_b;
__device__ int *ftr_SQ_f, *ftr_SQ_t;
__device__ int ftr_MQ_a[FTR_SIZE_S], ftr_MQ_b[FTR_SIZE_S];
__device__ int ftr_size_MQ_a, ftr_size_MQ_b;
__device__ int *ftr_MQ_f, *ftr_MQ_t;
__device__ int ftr_LQ_a[FTR_SIZE_S], ftr_LQ_b[FTR_SIZE_S];
__device__ int ftr_size_LQ_a, ftr_size_LQ_b;
__device__ int *ftr_LQ_f, *ftr_LQ_t;
__global__ void CUDA_BFS_KERNEL_CG_NEW(Node *node, int *edge, unsigned int *cost, bool *done, int *NUM_NODES)
{
    grid_group grid = this_grid();
    thread_group tile_S = tiled_partition(grid, TILE_SIZ_S);

    ftr_size_SQ_a = 0;
    ftr_size_SQ_b = 1;
    ftr_SQ_b[0] = 0; // TODO: SOURCE
    ftr_size_MQ_a = 0;
    ftr_size_MQ_b = 1;
    ftr_MQ_b[0] = 0; // TODO: SOURCE
    ftr_size_LQ_a = 0;
    ftr_size_LQ_b = 1;
    ftr_LQ_b[0] = 0; // TODO: SOURCE
    
    for(short iter = 0; !(*done) || !iter; iter++){
        if(threadIdx.x + blockIdx.x * blockDim.x == 0){
            // printf("iter: %d\nsize a/b: %d %d\n", iter, ftr_size_SQ_a, ftr_size_SQ_b);
            (iter % 2) ? (ftr_size_SQ_b = 0) : (ftr_size_SQ_a = 0);
            // printf("size a/b: %d %d\n", ftr_size_SQ_a, ftr_size_SQ_b);
            ftr_SQ_f = ((iter % 2) ? ftr_SQ_a : ftr_SQ_b);
            ftr_SQ_t = ((iter % 2) ? ftr_SQ_b : ftr_SQ_a);
        }
        grid.sync();
        *done = true;
        grid.sync();
        for(int j = ((threadIdx.x + blockIdx.x * blockDim.x) >> TILE_LOG_S); j < ((iter % 2) ? ftr_size_SQ_a : ftr_size_SQ_b); j += (blockDim.x * gridDim.x) >> TILE_LOG_S){
            // printf("%d %d\n", ftr_SQ_f[j], j);
            VISIT_tiled(tile_S, node, edge, cost, done, ftr_SQ_f[j], ftr_SQ_t, ((iter % 2) ? &ftr_size_SQ_b : &ftr_size_SQ_a));
        }
        grid.sync();
    }
}


// The BFS frontier corresponds to all the nodes being processed at the current level.



int main(int argc, char* argv[])
{
    printf("\033[0;1;33m"); cout << argv[1] + 15; printf("\033[0;1m\n");
	ifstream fin;
    int _, *NUM_NODES, NUM_EDGES, SOURCE;
    hipMallocManaged(&NUM_NODES, sizeof(int));
    fin.open(argv[1]);
    fin >> *NUM_NODES >> NUM_EDGES >> SOURCE;

	Node* node;
	int* edge;
	unsigned int* cost;
	int* count;
    hipMallocManaged(&node, sizeof(Node)*(*NUM_NODES));
    hipMallocManaged(&edge, sizeof(int)*NUM_EDGES);
    hipMallocManaged(&cost, sizeof(unsigned int)*(*NUM_NODES));
    hipMallocManaged(&count, sizeof(int));
    for(int i=0;i<*NUM_NODES;i++) fin >> node[i].start >> node[i].length;
    for(int i=0;i<NUM_EDGES;i++) fin >> edge[i] >> _;
    fin.close();

    for(string is_exit = ""; is_exit != "y"; )
    {
        do{
            if(is_exit == ""){
                cout << "# Nodes : " << *NUM_NODES << endl;
                cout << "# Edges : " <<  NUM_EDGES << endl;
                cout << "Source  : " <<     SOURCE << endl;
            }
            else{
                cout << "Source  : ";
                cin >> SOURCE;
            }
        } while(SOURCE >= *NUM_NODES || SOURCE < 0);

        for(int i=0;i<*NUM_NODES;i++) cost[i] = (i == SOURCE) ? 0 : 0xffffffff;

        vector<int> F;
        F.push_back(SOURCE);
        bool *X = (bool*)malloc(sizeof(bool)*(*NUM_NODES));
        for(int i=0;i<*NUM_NODES;i++) X[i] = (i == SOURCE);
        int *C = (int*)malloc(sizeof(int)*(*NUM_NODES));
        for(int i=0;i<*NUM_NODES;i++) C[i] = 0;
        // vector<vector<int>> frontier_every_round;
        // int maxi = 0;
        
        for(;!F.empty();)
        {
            vector<int> F_next;
            // maxi = F.size()>maxi?F.size():maxi;
            // vector<int> tmp;
            for(int i=F.size()-1;i>=0;i--)
            {
                int id = F[i];
                // tmp.push_back(id);
                int start = node[id].start;
                int end = start + node[id].length;
                for (int j = start; j < end; j++) 
                {
                    int nid = edge[j];
                    if (X[nid] == false)
                    {
                        X[nid] = true;
                        C[nid] = C[id] + 1;
                        F_next.push_back(nid);
                    }
                }
                F.pop_back();
            }
            F = F_next;
            // sort(tmp.begin(), tmp.end());
            // frontier_every_round.push_back(tmp);
        }

        int numBlocksPerSM = 1;
        int numThreads = 1024;
        bool* done;
        hipMallocManaged(&done, sizeof(bool));
        (*done) = false;
        (*count) = 0;
        int device = -1;
        hipGetDevice(&device);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSM, CUDA_BFS_KERNEL_CG_NEW, numThreads, 0);
        void *kernelArgs[] = {&node, &edge, &cost, &done, &NUM_NODES};
        dim3 num_blocks(deviceProp.multiProcessorCount * numBlocksPerSM, 1, 1);
        dim3 block_size(numThreads, 1, 1);
        // cout << "num of Nodes: " << *NUM_NODES << endl;
        // cout << "num_blocks: " << deviceProp.multiProcessorCount * numBlocksPerSM << endl;
        // cout << "block_size: " << numThreads << endl;

        hipMemPrefetchAsync(node, sizeof(Node)*(*NUM_NODES), device, NULL);
        hipMemPrefetchAsync(edge, sizeof(int)*NUM_EDGES, device, NULL);
        hipMemPrefetchAsync(cost, sizeof(unsigned int)*(*NUM_NODES), device, NULL);
        hipMemPrefetchAsync(done, sizeof(bool), device, NULL);
        hipMemPrefetchAsync(NUM_NODES, sizeof(int), device, NULL);
    
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        auto a = hipLaunchCooperativeKernel((void*)CUDA_BFS_KERNEL_CG_NEW, num_blocks, block_size, kernelArgs);
        hipDeviceSynchronize();
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float time;
        hipEventElapsedTime(&time, start, stop);

        int near_nodes = 0, near_errors = 0, dist = 0;
        for (int i = 0; i<*NUM_NODES; i++)
            if(X[i]){
                near_nodes++;
                if(cost[i] != C[i]) near_errors++;
                else dist = max(dist, cost[i]);
            }
        cout << "- Dist  = " << dist << endl;
        cout << "- Touch = " << (double)near_nodes / (double)(*NUM_NODES) * 100 << " % (" << near_nodes << " / " << *NUM_NODES << ")" << endl;
        (near_errors) ? printf("- Error = \033[31m%d\033[0;1m\n", near_errors) : printf("- Error = 0\n");
        cout << "- Time  = " << time << "ms" << endl;

        if(argv[argc - 1][0] == '$')
        {
            printf("\033[5mExit? [y/n]\033[0;1m ");
            cin >> is_exit;
            printf("\033[1A\033[K");
        }
        else is_exit = "y";
    }

    printf("\033[0m");
    hipFree(NUM_NODES);
    hipFree(node);
    hipFree(edge);
    hipFree(cost);
    hipFree(count);
}