#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <bits/stdc++.h>
#include <hip/hip_cooperative_groups.h>
using namespace std;
using namespace cooperative_groups;
namespace cg = cooperative_groups;

// shared-q/scan-free/non-sep/lower-bound/

#define TILE_LOG_A    0
#define TILE_SIZ_A    1
#define FTR_SIZE_A 6138

#define FTR_SIZE_G 16777216

typedef struct
{
	int start;     // Index of first adjacent node in Ea	
	int length;    // Number of adjacent nodes 
} Node;

__device__ inline void VISIT_tiled(thread_group tile, Node *node, int *edge, unsigned int *cost, bool *done, int ftr, int *ftr_AQ, int *ftr_size_AQ)
{
    int start = node[ftr].start;
    int end = start + node[ftr].length;
    
    for(int i = start+tile.thread_rank(); i < end; i += tile.size()){
        int nid = edge[i];
        if(atomicCAS(&(cost[nid]), (unsigned int)(0xffffffff), (unsigned int)(cost[ftr] + 1)) == (unsigned int)(0xffffffff)){
            if(node[nid].length) {
                ftr_AQ[atomicAdd(ftr_size_AQ, 1)] = nid;
            }
            *done = false;
        }
    }
}

// __device__ int NODE_i;
__device__ int ftr_GQ[FTR_SIZE_G], NODE_i;
__global__ void CUDA_BFS_KERNEL_CG_NEW(Node *node, int *edge, unsigned int *cost, bool *done, int *NUM_NODES)
{
    __shared__ int ftr_AQ_a[FTR_SIZE_A], ftr_size_AQ_a;
    __shared__ int ftr_AQ_b[FTR_SIZE_A], ftr_size_AQ_b;
    __shared__ int *ftr_AQ_f, *ftr_AQ_t, *ftr_size_AQ_f, *ftr_size_AQ_t, node_i;
    grid_group grid = this_grid();
    thread_group tile_S = tiled_partition(grid, TILE_SIZ_A);
    
    ftr_size_AQ_a = ftr_size_AQ_b = 0;
    __syncthreads();
    for(int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < *NUM_NODES; idx += gridDim.x * blockDim.x){
        if(!cost[idx]){
            NODE_i = 1;
            ftr_GQ[0] = idx;
        }
    }
    grid.sync();

    for(short iter = 0; !(*done) || !iter; iter++){
        if(threadIdx.x == 0){
            // printf("iter: %d\nsize a/b: %d %d\n", iter, ftr_size_SQ_a, ftr_size_SQ_b);
            ftr_size_AQ_f = ((iter % 2) ? &ftr_size_AQ_a : &ftr_size_AQ_b);
            ftr_size_AQ_t = ((iter % 2) ? &ftr_size_AQ_b : &ftr_size_AQ_a);
            *ftr_size_AQ_f = NODE_i / gridDim.x + (blockIdx.x < (NODE_i % gridDim.x));
            *ftr_size_AQ_t = 0;
            ftr_AQ_f = ((iter % 2) ? ftr_AQ_a : ftr_AQ_b);
            ftr_AQ_t = ((iter % 2) ? ftr_AQ_b : ftr_AQ_a);
            // node_i = blockIdx.x * blockDim.x;
        }
        __syncthreads();
        for(int i = threadIdx.x; i < *ftr_size_AQ_f; i += blockDim.x){
            ftr_AQ_f[i] = ftr_GQ[i + (NODE_i / gridDim.x) * blockIdx.x + ((blockIdx.x < (NODE_i % gridDim.x)) ? blockIdx.x : (NODE_i % gridDim.x))];
        }
        grid.sync();
        if(threadIdx.x + blockIdx.x * blockDim.x == 0){
            NODE_i = 0;// gridDim.x * blockDim.x;
        }
        *done = true;
        // while(node_i < *NUM_NODES){
        //     int idx = node_i + threadIdx.x;
        //     if(idx < *NUM_NODES && cost[idx] == iter && node[idx].length){
        //         ftr_AQ[atomicAdd(&ftr_size_AQ, 1)] = idx;
        //     }
        //     __syncthreads();
        //     if(ftr_size_AQ >= FTR_SIZE_A - blockDim.x){
        //         for(int j = threadIdx.x >> TILE_LOG_A; j < ((iter % 2) ? ftr_size_AQ_a : ftr_size_AQ_b); j += blockDim.x >> TILE_LOG_A){
        //             VISIT_tiled(tile_S, node, edge, cost, done, ftr_AQ_f[j], ftr_AQ_t, ((iter % 2) ? &ftr_size_AQ_b : &ftr_size_AQ_a));
        //         }
        //         __syncthreads();
        //         if(threadIdx.x == 0){
        //             ftr_size_AQ = 0;
        //         }
        //         __syncthreads();
        //     }
        //     if(threadIdx.x == 0){
        //         node_i = atomicAdd(&NODE_i, blockDim.x);
        //     }
        //     __syncthreads();
        // }
        grid.sync();
        for(int j = 0; j < *ftr_size_AQ_f; j += blockDim.x >> TILE_LOG_A){
            int idx = j + (threadIdx.x >> TILE_LOG_A);
            if(idx < *ftr_size_AQ_f){
                VISIT_tiled(tile_S, node, edge, cost, done, ftr_AQ_f[idx], ftr_AQ_t, ftr_size_AQ_t);
            }
            __syncthreads();
            if(threadIdx.x == 0){
                node_i = atomicAdd(&NODE_i, *ftr_size_AQ_t);
            }
            __syncthreads();
            for(int i = threadIdx.x; i < *ftr_size_AQ_t; i += blockDim.x){
                ftr_GQ[node_i + i] = ftr_AQ_t[i];
            }
            __syncthreads();
        }
        grid.sync();
    }
}

int main(int argc, char* argv[])
{
    printf("\033[0;1;33m"); cout << argv[1] + 15; printf("\033[0;1m\n");
	ifstream fin;
    int _, *NUM_NODES, NUM_EDGES, SOURCE;
    hipMallocManaged(&NUM_NODES, sizeof(int));
    fin.open(argv[1]);
    fin >> *NUM_NODES >> NUM_EDGES >> SOURCE;

	Node* node;
	int* edge;
	unsigned int* cost;
	int* count;
    hipMallocManaged(&node, sizeof(Node)*(*NUM_NODES));
    hipMallocManaged(&edge, sizeof(int)*NUM_EDGES);
    hipMallocManaged(&cost, sizeof(unsigned int)*(*NUM_NODES));
    hipMallocManaged(&count, sizeof(int));
    for(int i=0;i<*NUM_NODES;i++) fin >> node[i].start >> node[i].length;
    for(int i=0;i<NUM_EDGES;i++) fin >> edge[i] >> _;
    fin.close();

    for(string is_exit = ""; is_exit != "y"; )
    {
        do{
            if(is_exit == ""){
                cout << "# Nodes : " << *NUM_NODES << endl;
                cout << "# Edges : " <<  NUM_EDGES << endl;
                cout << "Source  : " <<     SOURCE << endl;
            }
            else{
                cout << "Source  : ";
                cin >> SOURCE;
            }
        } while(SOURCE >= *NUM_NODES || SOURCE < 0);

        for(int i=0;i<*NUM_NODES;i++) cost[i] = (i == SOURCE) ? 0 : 0xffffffff;

        vector<int> F;
        F.push_back(SOURCE);
        bool *X = (bool*)malloc(sizeof(bool)*(*NUM_NODES));
        for(int i=0;i<*NUM_NODES;i++) X[i] = (i == SOURCE);
        int *C = (int*)malloc(sizeof(int)*(*NUM_NODES));
        for(int i=0;i<*NUM_NODES;i++) C[i] = 0;
        // vector<vector<int>> frontier_every_round;
        // int maxi = 0;
        
        for(;!F.empty();)
        {
            vector<int> F_next;
            // maxi = F.size()>maxi?F.size():maxi;
            // vector<int> tmp;
            for(int i=F.size()-1;i>=0;i--)
            {
                int id = F[i];
                // tmp.push_back(id);
                int start = node[id].start;
                int end = start + node[id].length;
                for (int j = start; j < end; j++) 
                {
                    int nid = edge[j];
                    if (X[nid] == false)
                    {
                        X[nid] = true;
                        C[nid] = C[id] + 1;
                        F_next.push_back(nid);
                    }
                }
                F.pop_back();
            }
            F = F_next;
            // sort(tmp.begin(), tmp.end());
            // frontier_every_round.push_back(tmp);
        }

        int numBlocksPerSM = 1;
        int numThreads = 1024;
        bool* done;
        hipMallocManaged(&done, sizeof(bool));
        (*done) = false;
        (*count) = 0;
        int device = -1;
        hipGetDevice(&device);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSM, CUDA_BFS_KERNEL_CG_NEW, numThreads, 0);
        void *kernelArgs[] = {&node, &edge, &cost, &done, &NUM_NODES};
        dim3 num_blocks(deviceProp.multiProcessorCount * numBlocksPerSM, 1, 1);
        dim3 block_size(numThreads, 1, 1);
        // cout << "num of Nodes: " << *NUM_NODES << endl;
        // cout << "num_blocks: " << deviceProp.multiProcessorCount * numBlocksPerSM << endl;
        // cout << "block_size: " << numThreads << endl;

        hipMemPrefetchAsync(node, sizeof(Node)*(*NUM_NODES), device, NULL);
        hipMemPrefetchAsync(edge, sizeof(int)*NUM_EDGES, device, NULL);
        hipMemPrefetchAsync(cost, sizeof(unsigned int)*(*NUM_NODES), device, NULL);
        hipMemPrefetchAsync(done, sizeof(bool), device, NULL);
        hipMemPrefetchAsync(NUM_NODES, sizeof(int), device, NULL);
    
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        auto a = hipLaunchCooperativeKernel((void*)CUDA_BFS_KERNEL_CG_NEW, num_blocks, block_size, kernelArgs);
        hipDeviceSynchronize();
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float time;
        hipEventElapsedTime(&time, start, stop);

        int near_nodes = 0, near_errors = 0, dist = 0;
        for (int i = 0; i<*NUM_NODES; i++)
            if(X[i]){
                near_nodes++;
                if(cost[i] != C[i]) near_errors++;
                else dist = max(dist, cost[i]);
            }
        cout << "- Dist  = " << dist << endl;
        cout << "- Touch = " << (double)near_nodes / (double)(*NUM_NODES) * 100 << " % (" << near_nodes << " / " << *NUM_NODES << ")" << endl;
        (near_errors) ? printf("- Error = \033[31m%d\033[0;1m\n", near_errors) : printf("- Error = 0\n");
        cout << "- Time  = " << time << "ms" << endl;

        if(argv[argc - 1][0] == '$')
        {
            printf("\033[5mExit? [y/n]\033[0;1m ");
            cin >> is_exit;
            printf("\033[1A\033[K");
        }
        else is_exit = "y";
    }

    printf("\033[0m");
    hipFree(NUM_NODES);
    hipFree(node);
    hipFree(edge);
    hipFree(cost);
    hipFree(count);
}